#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include "stb_image.h"
#include "stb_image_write.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#define FILTRE_SIZE 3 


__global__ 
void PictureKernel (unsigned char* dPin, unsigned char* dPout, float *mask, int height, int width)
{	

	// Compute row and column number of dPin and dPout element
	const unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
	int position = row*width+col ;
	int up = position - width;
	int down = position + width;
	// Each thread computes one element of dPout if in range
	if(row==0||col==0||(row==width-1)||(row==height-1))
	{
		dPout[position] = dPin[position];
		return;
	}
	


	float val = (mask[0]*dPin[up-1] + mask[1]*dPin[up] + mask[2]*dPin[up + 1]
			+ mask[3]*dPin[position-1] + mask[4]*dPin[position] + mask[5]*dPin[position+1]
			+ mask[6]*dPin[down -1] + mask[7]*dPin[down] + mask[8]*dPin[down+1]);
	dPout[position] = (val <= 0 ? 0 : (val >= 255 ? 255 : (unsigned char)val));
}

void cuda_error(hipError_t err,const char *file,int line) {
	if (err != hipSuccess){
		printf("%s in %s at line %d\n" , hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
int main(void)
{
		int width = 0, height = 0, nchannels = 0;
		int const desired_channels = 1; // request to convert image to gray
		char const * const filename = "im.jpg"; 
	// Load the image 
	unsigned char* data_in = stbi_load(filename, &width, &height, &nchannels, desired_channels);

	// check for errors 
	if (!data_in || !width || !height || !nchannels){
	printf("Error loading image %s", filename);
	return -1;
	}

	// the filter  
	float mask[FILTRE_SIZE*FILTRE_SIZE] = { -1, -1, -1, -1, 8, -1, -1, -1, -1};


	unsigned char*data_out = (unsigned char*)malloc(width * height * desired_channels);

	// Memory allocation GPU
	unsigned char *gpu_data_in, *gpu_data_out;
	float * gpu_mask;
	
	cuda_error( hipMalloc (( void **)&gpu_data_in, width * height * desired_channels*sizeof(float)));
	
	cuda_error(hipMalloc (( void **)&gpu_data_out, width * height * desired_channels*sizeof(float)));
	cuda_error(hipMalloc (( void **)&gpu_mask, FILTRE_SIZE*FILTRE_SIZE*sizeof(float)));
	
	

	cuda_error(hipMemcpy (gpu_data_in, data_in, width * height * desired_channels*sizeof(float) , hipMemcpyHostToDevice));
	cuda_error(hipMemcpy (gpu_mask, mask , FILTRE_SIZE*FILTRE_SIZE*sizeof(float), hipMemcpyHostToDevice));
	


	// Set up the grid and block dimensions for the executions
	const unsigned int block_col = 16;
	const unsigned int block_row = 8;
	dim3 grid(height/block_col, width/ block_row, 1);
	dim3 threadBlock(block_col, block_row, 1);


		PictureKernel <<< grid, threadBlock >>>(gpu_data_in, gpu_data_out, gpu_mask, height, width);
	
	
	cuda_error(hipMemcpy (data_out, gpu_data_out, width * height * desired_channels, hipMemcpyDeviceToHost));


	stbi_write_jpg("sortie.jpg", height, width, 1, data_out, height);

	
	free(data_in);
	free(data_out);
	hipFree(gpu_data_in);
	hipFree(gpu_data_out);
	hipFree(gpu_mask);
	

}

