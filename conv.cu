#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define STB_IMAGE_IMPLEMENTATION

#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image.h"
#include "stb_image_write.h"
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <time.h>
#include <math.h>

#include "hip/hip_runtime.h"
#define FILTRE_SIZE 3 


__global__ 
void PictureKernel (unsigned char* dPin, unsigned char* dPout, float *mask, int height, int width)
{	

	// Compute row and column number of dPin and dPout element
	const unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
	int position = row*width+col ;
	int up = position - width;
	int down = position + width;
	// Each thread computes one element of dPout if in range
	if(row==0||col==0||(row==width-1)||(row==height-1))
	{
		dPout[position] = dPin[position];
		return;
	}
	


	float val = (mask[0]*dPin[up-1] + mask[1]*dPin[up] + mask[2]*dPin[up + 1]
			+ mask[3]*dPin[position-1] + mask[4]*dPin[position] + mask[5]*dPin[position+1]
			+ mask[6]*dPin[down -1] + mask[7]*dPin[down] + mask[8]*dPin[down+1]);
	dPout[position] = (val <= 0 ? 0 : (val >= 255 ? 255 : (unsigned char)val));
}

void cuda_error(hipError_t err,const char *file,int line) {
	if (err != hipSuccess){
		printf("%s in %s at line %d\n" , hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
int main(void)
{

		int width = 0, height = 0, nchannels = 0;
		int const desired_channels = 1; // request to convert image to gray
		char const * const filename = "im.jpg"; 
	// Load the image 
	unsigned char* data_in = stbi_load(filename, &width, &height, &nchannels, desired_channels);

	// check for errors 
	if (!data_in || !width || !height || !nchannels){
	printf("Error loading image %s", filename);
	return -1;
	}

	// the filter  
	float mask[FILTRE_SIZE*FILTRE_SIZE] = { -1, -1, -1, -1, 8, -1, -1, -1, -1};


	unsigned char*data_out = (unsigned char*)malloc(width * height * desired_channels);

	// Memory allocation GPU
	unsigned char *gpu_data_in, *gpu_data_out;
	float * gpu_mask;
	
	hipMalloc (( void **)&gpu_data_in, width * height * desired_channels*sizeof(float));
	hipMalloc (( void **)&gpu_data_out, width * height * desired_channels*sizeof(float));
	hipMalloc (( void **)&gpu_mask, FILTRE_SIZE*FILTRE_SIZE*sizeof(float));
	
	

	hipMemcpy (gpu_data_in, data_in, width * height * desired_channels*sizeof(float) , hipMemcpyHostToDevice);
	hipMemcpy (gpu_mask, mask , FILTRE_SIZE*FILTRE_SIZE*sizeof(float), hipMemcpyHostToDevice);
	


	// Set up the grid and block dimensions for the executions
	const unsigned int block_col = 32;
	const unsigned int block_row = 32;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	dim3 grid(height/block_col, width/ block_row, 1);
   	 dim3 threadBlock(block_col, block_row, 1);
  	hipEventRecord(start, 0);

	PictureKernel <<< grid, threadBlock >>>(gpu_data_in, gpu_mask,gpu_data_out,desired_channels,height, width);
	hipEventRecord(stop, 0);
	
	hipEventSynchronize(stop);
	float executionTime ;
	hipEventElapsedTime(&executionTime, start, stop);

	
	
	hipMemcpy (data_out, gpu_data_out, width * height * desired_channels, hipMemcpyDeviceToHost);
	
	//printf("SECTION 1 executionTime: %f", executionTime);
	//hipEventRecord(stop);
	stbi_write_jpg("sortie.jpg", height, width, 1, data_out, height);
	
	//  hipEventSynchronize(stop);
	
  	//float milliseconds = 0;
	
 	// hipEventElapsedTime(&milliseconds, start, stop);
	
       // printf("%f",milliseconds);
	free(data_in);
	free(data_out);
	hipFree(gpu_data_in);
	hipFree(gpu_data_out);
	hipFree(gpu_mask);
	

}
